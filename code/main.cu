#include "hip/hip_runtime.h"
#include "sparsemat.hpp"
#include "cuda_utils.hpp"
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <cassert>
#include <cstdio>
#include <iostream>
// #include <hip/hip_runtime_api.h>
// #include <helper_functions.h>

__device__ uint32_t MAX = ~0;

// sparse matrix multiplication C = A * B on the device 
// A is BCSR while B is BCSC
template <int m> __global__
void sparse_mul_cuda(uint32_t *A_data, int *A_idxs, int *A_idxptrs, 
        uint32_t *B_data, int *B_idxs, int *B_idxptrs, 
        uint32_t* C_data, uint8_t* C_valid, int n, int p) {


    // this block multiplies C(bx, by) <- sum_i A(bx, i) * B(i, by)
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // just return if there's nothing to do
    if (A_idxptrs[bx] == A_idxptrs[bx+1] || B_idxptrs[by] == B_idxptrs[by+1]) return;

    __shared__ uint32_t A_buf[m][m];
    __shared__ uint32_t B_buf[m][m];
    __shared__ uint64_t C_buf[m][m];

    C_buf[tx][ty] = 0;

    // use a two-pointer algorithm to find out which matrices to multiply first
    // invariants: 
    //     A_idxs[Ap] <= B_idxs[Bp]
    //     A_idxptrs[bx] <= Ap < A_idxptrs[bx+1]
    //     B_idxptrs[by] <= Bp < B_idxptrs[by+1]
    int Ap = A_idxptrs[bx];
    int Bp = B_idxptrs[by];

    uint8_t multiplied = 0;
    for (; Bp < B_idxptrs[by+1]; Bp++) {

        while (A_idxs[Ap] < B_idxs[Bp] && Ap < A_idxptrs[bx+1]) Ap++;
        if (Ap >= A_idxptrs[bx+1]) break;
        if (A_idxs[Ap] > B_idxs[Bp]) continue;

        // all invariants are satisfied here, so we can multiply
        multiplied = 1;

        // load into A_buf and B_buf 
        A_buf[tx][ty] = A_data[Ap*m*m + tx*m + ty];
        B_buf[tx][ty] = B_data[Bp*m*m + tx*m + ty];

        __syncthreads();
        // Do the multiplication
        #pragma unroll
        for (int i=0; i<m; i++) {
            C_buf[tx][ty] += (uint64_t)(A_buf[tx][i] * B_buf[i][ty]);
        }
    }

    if (!multiplied) return;

    // for (int i=0; i<m; i++) {
    //     for (int j=0; j<m; j++) {
    //         printf("%d ", C_buf[i][j]);
    //     }
    //     printf("\n");
    // }

    // copy C_buf to C_data: each thread copies one element
    if (tx == 0 && ty == 0) {
        // printf("Multiplied %d,%d\n", bx, by);
        C_valid[bx*p + by] = 1;
    }
    __syncthreads();
    if (C_buf[tx][ty] >> 32) C_data[(bx*m+tx)*n + by*m + ty] = MAX;
    else C_data[(bx*m + tx)*n + by*m + ty] = (uint32_t)C_buf[tx][ty];
}

__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__
void marginalize_rows(uint8_t* C_valid, int* C_rowsums, int p) {

    int rid = blockIdx.x;
    int tid = threadIdx.x;
    // block has 256 threads
    __shared__ int mergesums[256];
    mergesums[tid] = 0;

    // reduce first to a 256-len array
    int i = tid;
    while (i < p) {
        mergesums[tid] += (int)C_valid[rid*p + i];
        i += 256;
    }
    __syncthreads();

    // now reduce mergesums
    if (tid < 128) mergesums[tid] += mergesums[tid+128];
    __syncthreads();
    if (tid < 64) mergesums[tid] += mergesums[tid+64];
    __syncthreads();

    if (tid < 32) warpReduce(mergesums, tid);
    __syncthreads();

    if (tid == 0) {
        C_rowsums[rid+1] = mergesums[0];
    }
}

__global__
void compress_data(uint8_t* C_valid, int* C_idxptrs, int* C_idxs, 
        uint32_t* C_data, uint32_t* C_bcsr_data, int n, int m, int p) {

    int row = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ int idx;
    
    if (tx == 0 && ty == 0) idx = C_idxptrs[row];
    __syncthreads();

    for (int i=0; i<p; i++) {
        if (C_valid[p*row + i] == 1) {
            // all threads compress C_data
            C_bcsr_data[idx*m*m + tx*m + ty] = C_data[row*n*m + tx*n + i*m + ty];
            __syncthreads();
            if (tx == 0 && ty == 0) {
                C_idxs[idx] = i;
                idx++;
            }
            __syncthreads();
        }
    }
}

BCSMatrix* sparse_matrix_multiply(BCSMatrix *A, BCSMatrix *B) {

    // allocate device memory

    // run the kernel on the device
    // n is upto 2^15, m is either 2^2 or 2^3 so n/m <= 2^13
    // kernel supports 2^16 block size tops, so it's easy to parallelize 
    // everything all at once. If a block is empty, then we'll just leave
    // it out.

    // setup a stream for the matrix. Initialize d_A,d_B,d_C on the device and
    // then copy over essential information first, then move the actual matrix
    // over...

    // use thrust maps to figure out which blocks are along which coordinates.
    // Store these maps on the matrix itself to make multiplication faster.
    // 
    // NOPE, thrust doesn't have maps... we need to use a CSR/CSC mat for that.
    // each block will be responsible for computing all elements of that block.

    // how do we bookkeep? we'll need to mark out and allocate space for empty 
    // blocks and add the indices to the final CSR matrix as well. Don't need
    // conflicts here. Can do this in the final synchronize loop.
    // maybe can try using a thrust vector here for C

    // DCSR may be one option
    // given the constraints, I think the easiest option is to allocate a 
    // large 2-D map and then parallelly allocate pointers on it.
    // Then reduce the map in parallel 

    assert(A->ct == CT_ROW);
    assert(B->ct == CT_COL);

    // need to think of device memory allocation... Let's just assume that we 
    // get 12GB of device memory when we're running, so hipMallocManaged 
    // shouldn't be an issue...

    uint32_t *d_A_data,    *d_B_data;
    int *d_A_idxs,    *d_B_idxs;
    int *d_A_idxptrs, *d_B_idxptrs;

    uint32_t* d_C_data;
    int* d_C_idxptrs;
    uint8_t*  d_C_valid;
    //uint32_t* h_C_data;


    // create streams
    // hipStream_t stream;
    // checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // assuming A->m = B->m = m
    int p = A->p;
    int m = A->m;
    int n = A->n;

    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A_idxptrs), sizeof(int)*(A->p+1)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A_idxs),    sizeof(int)*A->k));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A_data),    sizeof(uint32_t)*A->k*A->m*A->m));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B_idxptrs), sizeof(int)*(B->p+1)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B_idxs),    sizeof(int)*B->k));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B_data),    sizeof(uint32_t)*B->k*B->m*B->m));

    // checkCudaErrors(hipMemcpyAsync(d_A_idxptrs, A->idxptrs, sizeof(int)*(A->p+1), hipMemcpyHostToDevice, stream));
    // checkCudaErrors(hipMemcpyAsync(d_A_idxs,    A->idxs,    sizeof(int)*A->k, hipMemcpyHostToDevice, stream));
    // checkCudaErrors(hipMemcpyAsync(d_A_data,    A->data,    sizeof(uint32_t)*A->k*A->m*A->m, hipMemcpyHostToDevice, stream));
    // checkCudaErrors(hipMemcpyAsync(d_B_idxptrs, B->idxptrs, sizeof(int)*(B->p+1), hipMemcpyHostToDevice, stream));
    // checkCudaErrors(hipMemcpyAsync(d_B_idxs,    B->idxs,    sizeof(int)*B->k, hipMemcpyHostToDevice, stream));
    // checkCudaErrors(hipMemcpyAsync(d_B_data,    B->data,    sizeof(uint32_t)*B->k*B->m*B->m, hipMemcpyHostToDevice, stream));

    checkCudaErrors(hipMemcpy(d_A_idxptrs, A->idxptrs, sizeof(int)*(A->p+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_A_idxs,    A->idxs,    sizeof(int)*A->k, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_A_data,    A->data,    sizeof(uint32_t)*A->k*A->m*A->m, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B_idxptrs, B->idxptrs, sizeof(int)*(B->p+1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B_idxs,    B->idxs,    sizeof(int)*B->k, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B_data,    B->data,    sizeof(uint32_t)*B->k*B->m*B->m, hipMemcpyHostToDevice));

    // initialize C matrix on device
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C_data),  n*n*sizeof(uint32_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C_valid), p*p*sizeof(uint8_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C_idxptrs), (p+1)*sizeof(int)));
    // checkCudaErrors(hipHostMalloc(reinterpret_cast<void**>(&h_C_data),  n*n*sizeof(uint32_t)));
    // JUST WHILE TESTING!
    // checkCudaErrors(hipMemset(reinterpret_cast<void*>(d_C_data), 0, n*n*sizeof(uint32_t)));
    checkCudaErrors(hipMemset(reinterpret_cast<void*>(d_C_valid), 0, p*p*sizeof(uint8_t)));

    dim3 threads(m, m);
    dim3 grid(p, p);

    // checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipDeviceSynchronize());

    // now multiply the stuff out
    if (A->m == 4) {
        sparse_mul_cuda<4><<<grid, threads>>>(
            d_A_data, d_A_idxs, d_A_idxptrs, d_B_data, d_B_idxs, d_B_idxptrs,
            d_C_data, d_C_valid, n, p
        );
    }
    else {
        sparse_mul_cuda<8><<<grid, threads>>>(
            d_A_data, d_A_idxs, d_A_idxptrs, d_B_data, d_B_idxs, d_B_idxptrs,
            d_C_data, d_C_valid, n, p
        );
    }
    checkCudaErrors(hipDeviceSynchronize());

    // checkCudaErrors(hipStreamSynchronize(stream));
    // checkCudaErrors(hipMemcpyAsync(h_C_data,  d_C_data,  n*n*sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
    // checkCudaErrors(hipStreamSynchronize(stream));
    // for (int i=0; i<n; i++) {
    //     for (int j=0; j<n; j++) {
    //         std::cout << h_C_data[i*n+j] << ' ';
    //     }
    //     std::cout << '\n';
    // }

    // uint8_t *h_C_valid;
    // checkCudaErrors(hipHostMalloc(reinterpret_cast<void**>(&h_C_valid), p*p*sizeof(uint8_t)));
    // checkCudaErrors(hipMemcpy(h_C_valid, d_C_valid, p*p*sizeof(uint8_t), hipMemcpyDeviceToHost));
    // checkCudaErrors(hipStreamSynchronize(stream));
    // for (int i=0; i<p; i++) {
    //     for (int j=0; j<p; j++) {
    //         std::cout << (int)h_C_valid[i*p+j] << ' ';
    //     }
    //     std::cout << '\n';
    // }

    // marginalize the rows out
    marginalize_rows<<<p,256>>>(d_C_valid, d_C_idxptrs, p);
    checkCudaErrors(hipDeviceSynchronize());
    // checkCudaErrors(hipStreamSynchronize(stream));
    // sum them up inplace to get the start pointers
    thrust::device_ptr<int> thrust_d_C_idxptrs(d_C_idxptrs);
    thrust::inclusive_scan(thrust_d_C_idxptrs, thrust_d_C_idxptrs+p+1, thrust_d_C_idxptrs);

    int *C_idxptrs;
    checkCudaErrors(hipHostMalloc(reinterpret_cast<void**>(&C_idxptrs), (p+1)*sizeof(int)));
    checkCudaErrors(hipMemcpy(C_idxptrs, d_C_idxptrs, (p+1)*sizeof(int), hipMemcpyDeviceToHost));

    // allocate memory for obtaining row indexes from the last value of d_C_idxptrs
    int *C_idxs, *d_C_idxs;
    uint32_t *C_bcsr_data;
    uint32_t *d_C_bcsr_data;

    int k = C_idxptrs[p];
    
    checkCudaErrors(hipHostMalloc(reinterpret_cast<void**>(&C_idxs), k*sizeof(int)));
    checkCudaErrors(hipHostMalloc(reinterpret_cast<void**>(&C_bcsr_data), k*m*m*sizeof(uint32_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C_bcsr_data), k*m*m*sizeof(uint32_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C_idxs), k*sizeof(int)));

    compress_data<<<p, threads>>>(d_C_valid, d_C_idxptrs, d_C_idxs, d_C_data, d_C_bcsr_data, n, m, p);
    // checkCudaErrors(hipStreamSynchronize(stream));

    checkCudaErrors(hipMemcpy(C_idxs, d_C_idxs, k*sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(C_bcsr_data, d_C_bcsr_data, k*m*m*sizeof(int), hipMemcpyDeviceToHost));
    // checkCudaErrors(hipMemcpyAsync(C_idxs, d_C_idxs, k*sizeof(int), hipMemcpyDeviceToHost, stream));
    // checkCudaErrors(hipMemcpyAsync(C_bcsr_data, d_C_bcsr_data, k*m*m*sizeof(int), hipMemcpyDeviceToHost, stream));
    // checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipDeviceSynchronize());

    // TODO compress the C matrix into CSR format
    // for now, we can just copy it over

    checkCudaErrors(hipFree(d_A_idxptrs));
    checkCudaErrors(hipFree(d_A_idxs));
    checkCudaErrors(hipFree(d_A_data));
    checkCudaErrors(hipFree(d_B_idxptrs));
    checkCudaErrors(hipFree(d_B_idxs));
    checkCudaErrors(hipFree(d_B_data));
    checkCudaErrors(hipFree(d_C_idxptrs));
    checkCudaErrors(hipFree(d_C_idxs));
    checkCudaErrors(hipFree(d_C_data));
    checkCudaErrors(hipFree(d_C_bcsr_data));
    checkCudaErrors(hipFree(d_C_valid));
    // checkCudaErrors(hipHostFree(h_C_valid));

    BCSMatrix *C = new BCSMatrix(n, m, k, CT_ROW);
    C->idxptrs = C_idxptrs;
    C->idxs = C_idxs;
    C->data = C_bcsr_data;

    return C;
}

int main(int argc, char** argv) {

    BCSMatrix* A = new BCSMatrix(argv[1], CT_ROW);
    BCSMatrix* B = new BCSMatrix(argv[2], CT_COL);

    // A->print();
    // A->dense_print();
    // std::cout << '\n';
    // B->print();
    // B->dense_print();
    // std::cout << '\n';

    BCSMatrix *C = sparse_matrix_multiply(A, B);

    checkCudaErrors(hipDeviceSynchronize());

    // C->print();
    // C->dense_print();
    // std::cout << '\n';

    C->save(argv[3]);

    delete A;
    delete B;
    delete C;

    return 0;
}

