#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void test(){
    printf("Hi Cuda World\n");
}

int main( int argc, char** argv )
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
